#ifndef WINDOWS
 #include <unistd.h>
#endif

#include <stdio.h>

#include <hip/hip_runtime.h>

#define GPU_DEVICE 0

int main(int argc, char *argv[])
{
  int devID = 0;
  hipError_t error;
  hipDeviceProp_t deviceProp;
  error = hipGetDevice(&devID);

  int runtimeVersion=0;
  int driverVersion=0;

  error=hipRuntimeGetVersion(&runtimeVersion);
  if (error == hipSuccess) {
    printf("CUDA runtime version: %d\n", runtimeVersion);
  }

  error=hipDriverGetVersion(&driverVersion);
  if (error == hipSuccess) {
    printf("CUDA driver version: %d\n", driverVersion);
  }

  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  if (error == hipSuccess) {
    printf("GPU Device ID: %d\n", devID);
    printf("GPU Name: %s\n", deviceProp.name);
    printf("GPU compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
  }
  else {
    printf("Can't initialize CUDA device, return code: %d\n", error);
  }

  return error;
}
