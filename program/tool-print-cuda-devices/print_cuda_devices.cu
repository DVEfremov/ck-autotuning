#ifndef WINDOWS
 #include <unistd.h>
#endif

#include <stdio.h>

#include <hip/hip_runtime.h>

#define GPU_DEVICE 0

int main(int argc, char *argv[])
{
  int devID = 0;
  hipError_t error;
  hipDeviceProp_t deviceProp;
  error = hipGetDevice(&devID);

  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  if (error == hipSuccess) {
    printf("GPU Device ID: %d\n", devID);
    printf("GPU Name: %s\n", deviceProp.name);
    printf("GPU compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
  }
  else {
    printf("Can't initialize CUDA device, return code: %d\n", error);
  }

  return error;
}
